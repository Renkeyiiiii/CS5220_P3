#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "../common/common.hpp"
#include "../common/solver.hpp"
#include <stdio.h>


// Global variables for device memory
double *d_h, *d_u, *d_v;           // Current fields
double *d_dh, *d_du, *d_dv;        // Current derivatives
double *d_dh1, *d_du1, *d_dv1;     // Previous derivatives
double *d_dh2, *d_du2, *d_dv2;     // Two steps ago derivatives

// Constants for the simulation
int nx, ny;
double H, g, dx, dy, dt;
int t = 0;

__global__ void compute_ghost_kernel(double *h, int nx, int ny)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // int stride = blockDim.x * gridDim.x;
    int i = index;

    if (index < ny)
    {

            h(nx, i) = h(0, i);

    }
    else if (index < nx + ny)
    {

            h(i-ny, ny) = h(i-ny, 0);

    }
}

__global__ void compute_dh_kernel(double *h, double *u, double *v, double *dh, 
                                int nx, int ny, double H, double dx, double dy) {

    int index_col = blockIdx.x * blockDim.x + threadIdx.x;
    int index_row = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_col = blockDim.x * gridDim.x;
    int stride_row = blockDim.y * gridDim.y;
    for (int i = index_col; i < nx; i += stride_col)
    {
        for (int j = index_row; j < ny; j += stride_row)
        {
            dh(i, j) = -H * (du_dx(i, j) + dv_dy(i, j));
        }
    }
}

__global__ void compute_du_kernel(double *h, double *du, 
                                int nx, int ny, double g, double dx) {

    int index_col = blockIdx.x * blockDim.x + threadIdx.x;
    int index_row = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_col = blockDim.x * gridDim.x;
    int stride_row = blockDim.y * gridDim.y;
    for (int i = index_col; i < nx; i += stride_col)
    {
        for (int j = index_row; j < ny; j += stride_row)
        {
            du(i, j) = -g * dh_dx(i, j);
        }
    }
}

__global__ void compute_dv_kernel(double *h, double *dv, 
                                int nx, int ny, double g, double dy) {

    int index_col = blockIdx.x * blockDim.x + threadIdx.x;
    int index_row = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_col = blockDim.x * gridDim.x;
    int stride_row = blockDim.y * gridDim.y;
    for (int i = index_col; i < nx; i += stride_col)
    {
        for (int j = index_row; j < ny; j += stride_row)
        {
            dv(i, j) = -g * dh_dy(i, j);
        }
    }
}

__global__ void multistep_kernel(double *h, double *u, double *v,
                                double *dh, double *du, double *dv,
                                double *dh1, double *du1, double *dv1,
                                double *dh2, double *du2, double *dv2,
                                int nx, int ny, double dt,
                                double a1, double a2, double a3) 
{

    int index_col = blockIdx.x * blockDim.x + threadIdx.x;
    int index_row = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_col = blockDim.x * gridDim.x;
    int stride_row = blockDim.y * gridDim.y;
    
    for (int i = index_col; i < nx; i += stride_col)
    {
        for (int j = index_row; j < ny; j += stride_row)
        {
            h(i, j) += (a1 * dh(i, j) + a2 * dh1(i, j) + a3 * dh2(i, j)) * dt;
            u(i + 1, j) += (a1 * du(i, j) + a2 * du1(i, j) + a3 * du2(i, j)) * dt;
            v(i, j + 1) += (a1 * dv(i, j) + a2 * dv1(i, j) + a3 * dv2(i, j)) * dt;
        }
    }
}


// We are going to be doing four things here:
// 1.compute_dh
// 2.compute_du
// 3.compute_dv
// 4.multistep
__global__ void new_kernel(double *h, double *u, double *v,
                                double *dh, double *du, double *dv,
                                double *dh1, double *du1, double *dv1,
                                double *dh2, double *du2, double *dv2,
                                int nx, int ny, double dt,
                                double H, double g, double dx, double dy,
                                int t) 
{ 
    int index_col = blockIdx.x * blockDim.x + threadIdx.x;
    int index_row = blockIdx.y * blockDim.y + threadIdx.y;
    int i = index_col;
    int j = index_row;

    if(i < nx && j < ny)
    {
        dh(i, j) = -H * (du_dx(i, j) + dv_dy(i, j));
        du(i, j) = -g * dh_dx(i, j);
        dv(i, j) = -g * dh_dy(i, j);
    }

    __syncthreads();
    // test
    if (t == 100 && i == 0 & j == 0)
    {
        printf("dh(0,0) = %lf\n", dh(0,0));
        printf("du(0,0) = %lf\n", du(0,0));
        printf("dv(0,0) = %lf\n", dv(0,0));
        printf("du_dx(0,0) = %lf\n", du_dx(0,0));
        printf("dv_dy(0,0) = %lf\n", dv_dy(0,0));
        printf("dh_dx(0,0) = %lf\n", dh_dx(0,0));
        printf("dh_dy(0,0) = %lf\n", dh_dy(0,0));
        printf("h(0,0) = %lf h(1,0) = %lf h(0,1) = %lf\n", h(0,0), h(1,0), h(0,1));
        printf("u(0,0) = %lf u(1,0) = %lf\n", u(0,0), u(1,0));
        printf("v(0,0) = %lf v(0,1) = %lf\n", v(0,0), v(0,1));
    }
    __syncthreads();
    double a1, a2 = 0.0, a3 = 0.0;
    if (t == 0) {
        a1 = 1.0;
    } else if (t == 1) {
        a1 = 3.0 / 2.0;
        a2 = -1.0 / 2.0;
    } else {
        a1 = 23.0 / 12.0;
        a2 = -16.0 / 12.0;
        a3 = 5.0 / 12.0;
    } 

    if(i < nx && j < ny)
    {
        h(i, j) += (a1 * dh(i, j) + a2 * dh1(i, j) + a3 * dh2(i, j)) * dt;
        u(i + 1, j) += (a1 * du(i, j) + a2 * du1(i, j) + a3 * du2(i, j)) * dt;
        v(i, j + 1) += (a1 * dv(i, j) + a2 * dv1(i, j) + a3 * dv2(i, j)) * dt;
    }
}

__global__ void compute_boundaries_kernel(double *h, double *u, double *v, int nx, int ny) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // int stride = blockDim.x * gridDim.x;
    int i = index;

    if (index < ny)
    {

            u(0, i) = u(nx, i);

    }
    else if (index < nx + ny)
    {

            v(i-ny, 0) = v(i-ny, ny);

    }
}

void init(double *h0, double *u0, double *v0, double length_, double width_, 
          int nx_, int ny_, double H_, double g_, double dt_, int rank_, int num_procs_) {
    // Store grid dimensions and parameters
    nx = nx_;
    ny = ny_;
    H = H_;
    g = g_;
    dx = length_ / nx;
    dy = width_ / ny;
    dt = dt_;
    
    // Allocate device memory with correct sizes
    hipMalloc(&d_h, (nx + 1) * (ny + 1) * sizeof(double));  // h has extra points in both directions
    hipMalloc(&d_u, (nx + 2) * ny * sizeof(double));        // u has extra points in x direction
    hipMalloc(&d_v, nx * (ny + 2) * sizeof(double));        // v has extra points in y direction
    
    // Allocate device memory for derivatives
    size_t deriv_size = nx * ny * sizeof(double);
    hipMalloc(&d_dh, deriv_size);
    hipMalloc(&d_du, deriv_size);
    hipMalloc(&d_dv, deriv_size);
    hipMalloc(&d_dh1, deriv_size);
    hipMalloc(&d_du1, deriv_size);
    hipMalloc(&d_dv1, deriv_size);
    hipMalloc(&d_dh2, deriv_size);
    hipMalloc(&d_du2, deriv_size);
    hipMalloc(&d_dv2, deriv_size);
    
    // Copy initial conditions to device with correct sizes
    hipMemcpy(d_h, h0, (nx + 1) * (ny + 1) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_u, u0, (nx + 2) * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v0, nx * (ny + 2) * sizeof(double), hipMemcpyHostToDevice);
    
    // Initialize derivative arrays to zero
    hipMemset(d_dh, 0, deriv_size);
    hipMemset(d_du, 0, deriv_size);
    hipMemset(d_dv, 0, deriv_size);
    hipMemset(d_dh1, 0, deriv_size);
    hipMemset(d_du1, 0, deriv_size);
    hipMemset(d_dv1, 0, deriv_size);
    hipMemset(d_dh2, 0, deriv_size);
    hipMemset(d_du2, 0, deriv_size);
    hipMemset(d_dv2, 0, deriv_size);
}

void step() {
    // Define block and grid dimensions
    dim3 block(16, 16);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    dim3 boundary_block(256);
    dim3 boundary_grid(((nx + ny) + boundary_block.x - 1) / boundary_block.x);

    compute_ghost_kernel<<<boundary_grid, boundary_block>>>(d_h, nx, ny);
    hipDeviceSynchronize();
    // Compute derivatives

    // compute_dh_kernel<<<grid, block>>>(d_h, d_u, d_v, d_dh, nx, ny, H, dx, dy);
    // hipDeviceSynchronize();
    // compute_du_kernel<<<grid, block>>>(d_h, d_du, nx, ny, g, dx);
    // hipDeviceSynchronize();
    // compute_dv_kernel<<<grid, block>>>(d_h, d_dv, nx, ny, g, dy);
    // hipDeviceSynchronize();

    new_kernel<<<grid, block>>>(d_h, d_u, d_v, d_dh, d_du, d_dv,
                        d_dh1, d_du1, d_dv1, d_dh2, d_du2, d_dv2,
                        nx, ny, dt, H, g, dx, dy, t);
    
    // Set multistep coefficients
    // double a1, a2 = 0.0, a3 = 0.0;
    // if (t == 0) {
    //     a1 = 1.0;
    // } else if (t == 1) {
    //     a1 = 3.0 / 2.0;
    //     a2 = -1.0 / 2.0;
    // } else {
    //     a1 = 23.0 / 12.0;
    //     a2 = -16.0 / 12.0;
    //     a3 = 5.0 / 12.0;
    // }
    
    // Update fields
    // multistep_kernel<<<grid, block>>>(d_h, d_u, d_v, d_dh, d_du, d_dv,
    //                                  d_dh1, d_du1, d_dv1, d_dh2, d_du2, d_dv2,
    //                                  nx, ny, dt, a1, a2, a3);
    // hipDeviceSynchronize();
    // Handle boundaries
    compute_boundaries_kernel<<<boundary_grid, boundary_block>>>(d_h, d_u, d_v, nx, ny);

    hipDeviceSynchronize();
    
    // Swap derivative buffers
    double *tmp;
    tmp = d_dh2; d_dh2 = d_dh1; d_dh1 = d_dh; d_dh = tmp;
    tmp = d_du2; d_du2 = d_du1; d_du1 = d_du; d_du = tmp;
    tmp = d_dv2; d_dv2 = d_dv1; d_dv1 = d_dv; d_dv = tmp;
    
    t++;
}

void transfer(double *h_host) {
    hipMemcpy(h_host, d_h, (nx + 1) * (ny + 1) * sizeof(double), hipMemcpyDeviceToHost);
}

void free_memory() {
    hipFree(d_h);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_dh);
    hipFree(d_du);
    hipFree(d_dv);
    hipFree(d_dh1);
    hipFree(d_du1);
    hipFree(d_dv1);
    hipFree(d_dh2);
    hipFree(d_du2);
    hipFree(d_dv2);
}