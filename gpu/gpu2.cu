#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "../common/common.hpp"
#include "../common/solver.hpp"


// Global variables for device memory
double *d_h, *d_u, *d_v;           // Current fields
double *d_dh, *d_du, *d_dv;        // Current derivatives
double *d_dh1, *d_du1, *d_dv1;     // Previous derivatives
double *d_dh2, *d_du2, *d_dv2;     // Two steps ago derivatives

// Constants for the simulation
int nx, ny;
double H, g, dx, dy, dt;
int t = 0;

__global__ void compute_dh_kernel(double *h, double *u, double *v, double *dh, 
                                int nx, int ny, double H, double dx, double dy) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < nx && j < ny) {
        // Using same macros as serial code
        dh(i, j) = -H * (du_dx(i, j) + dv_dy(i, j));
    }
}

__global__ void compute_du_kernel(double *h, double *du, 
                                int nx, int ny, double g, double dx) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < nx && j < ny) {
        du(i, j) = -g * dh_dx(i, j);
    }
}

__global__ void compute_dv_kernel(double *h, double *dv, 
                                int nx, int ny, double g, double dy) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < nx && j < ny) {
        dv(i, j) = -g * dh_dy(i, j);
    }
}

__global__ void multistep_kernel(double *h, double *u, double *v,
                                double *dh, double *du, double *dv,
                                double *dh1, double *du1, double *dv1,
                                double *dh2, double *du2, double *dv2,
                                int nx, int ny, double dt,
                                double a1, double a2, double a3) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < nx && j < ny) {
        // Update height field
        h(i, j) += (a1 * dh(i, j) + a2 * dh1(i, j) + a3 * dh2(i, j)) * dt;
        
        // Update velocity fields
        if (i < nx-1) {
            u(i+1, j) += (a1 * du(i, j) + a2 * du1(i, j) + a3 * du2(i, j)) * dt;
        }
        if (j < ny-1) {
            v(i, j+1) += (a1 * dv(i, j) + a2 * dv1(i, j) + a3 * dv2(i, j)) * dt;
        }
    }
}

__global__ void compute_boundaries_kernel(double *h, double *u, double *v, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Horizontal boundaries
    if (idx < ny) {
        h(nx, idx) = h(0, idx);
        u(0, idx) = u(nx, idx);
    }
    
    // Vertical boundaries
    if (idx < nx) {
        h(idx, ny) = h(idx, 0);
        v(idx, 0) = v(idx, ny);
    }
}

void init(double *h0, double *u0, double *v0, double length_, double width_, 
          int nx_, int ny_, double H_, double g_, double dt_, int rank_, int num_procs_) {
    // Store grid dimensions and parameters
    nx = nx_;
    ny = ny_;
    H = H_;
    g = g_;
    dx = length_ / nx;
    dy = width_ / ny;
    dt = dt_;
    
    // Allocate device memory with correct sizes
    hipMalloc(&d_h, (nx + 1) * (ny + 1) * sizeof(double));  // h has extra points in both directions
    hipMalloc(&d_u, (nx + 2) * ny * sizeof(double));        // u has extra points in x direction
    hipMalloc(&d_v, nx * (ny + 2) * sizeof(double));        // v has extra points in y direction
    
    // Allocate device memory for derivatives
    size_t deriv_size = nx * ny * sizeof(double);
    hipMalloc(&d_dh, deriv_size);
    hipMalloc(&d_du, deriv_size);
    hipMalloc(&d_dv, deriv_size);
    hipMalloc(&d_dh1, deriv_size);
    hipMalloc(&d_du1, deriv_size);
    hipMalloc(&d_dv1, deriv_size);
    hipMalloc(&d_dh2, deriv_size);
    hipMalloc(&d_du2, deriv_size);
    hipMalloc(&d_dv2, deriv_size);
    
    // Copy initial conditions to device with correct sizes
    hipMemcpy(d_h, h0, (nx + 1) * (ny + 1) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_u, u0, (nx + 2) * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v0, nx * (ny + 2) * sizeof(double), hipMemcpyHostToDevice);
    
    // Initialize derivative arrays to zero
    hipMemset(d_dh, 0, deriv_size);
    hipMemset(d_du, 0, deriv_size);
    hipMemset(d_dv, 0, deriv_size);
    hipMemset(d_dh1, 0, deriv_size);
    hipMemset(d_du1, 0, deriv_size);
    hipMemset(d_dv1, 0, deriv_size);
    hipMemset(d_dh2, 0, deriv_size);
    hipMemset(d_du2, 0, deriv_size);
    hipMemset(d_dv2, 0, deriv_size);
}

void step() {
    // Define block and grid dimensions
    dim3 block(16, 16);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    
    // Compute derivatives
    compute_dh_kernel<<<grid, block>>>(d_h, d_u, d_v, d_dh, nx, ny, H, dx, dy);
    compute_du_kernel<<<grid, block>>>(d_h, d_du, nx, ny, g, dx);
    compute_dv_kernel<<<grid, block>>>(d_h, d_dv, nx, ny, g, dy);
    
    // Set multistep coefficients
    double a1, a2 = 0.0, a3 = 0.0;
    if (t == 0) {
        a1 = 1.0;
    } else if (t == 1) {
        a1 = 3.0 / 2.0;
        a2 = -1.0 / 2.0;
    } else {
        a1 = 23.0 / 12.0;
        a2 = -16.0 / 12.0;
        a3 = 5.0 / 12.0;
    }
    
    // Update fields
    multistep_kernel<<<grid, block>>>(d_h, d_u, d_v, d_dh, d_du, d_dv,
                                     d_dh1, d_du1, d_dv1, d_dh2, d_du2, d_dv2,
                                     nx, ny, dt, a1, a2, a3);
    
    // Handle boundaries
    dim3 boundary_block(256);
    dim3 boundary_grid((max(nx, ny) + boundary_block.x - 1) / boundary_block.x);
    compute_boundaries_kernel<<<boundary_grid, boundary_block>>>(d_h, d_u, d_v, nx, ny);
    
    // Swap derivative buffers
    double *tmp;
    tmp = d_dh2; d_dh2 = d_dh1; d_dh1 = d_dh; d_dh = tmp;
    tmp = d_du2; d_du2 = d_du1; d_du1 = d_du; d_du = tmp;
    tmp = d_dv2; d_dv2 = d_dv1; d_dv1 = d_dv; d_dv = tmp;
    
    t++;
}

void transfer(double *h_host) {
    hipMemcpy(h_host, d_h, (nx + 1) * (ny + 1) * sizeof(double), hipMemcpyDeviceToHost);
}

void free_memory() {
    hipFree(d_h);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_dh);
    hipFree(d_du);
    hipFree(d_dv);
    hipFree(d_dh1);
    hipFree(d_du1);
    hipFree(d_dv1);
    hipFree(d_dh2);
    hipFree(d_du2);
    hipFree(d_dv2);
}